#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <stdlib.h>


// Defining number of elements in Array
#define N 5

// Defining Kernel function for vector addition
__global__ void gpuAdd(int *d_a, int *d_b, int *d_c)
{
    // Getting block index of current kernel
    int tid = blockIdx.x; // handle the data at this index
    if (tid < N)
        d_c[tid] = d_a[tid] + d_b[tid];
}

int main(void)
{
    // Defining host arrays
    int h_a[N], h_b[N], h_c[N];
    // Defining device pointers
    int *d_a, *d_b, *d_c;
    // allocate the memory
    hipMalloc((void**)&d_a, N * sizeof(int));
    hipMalloc((void**)&d_b, N * sizeof(int));
    hipMalloc((void**)&d_c, N * sizeof(int));
    // Initializing Arrays
    for (int i = 0; i < N; i++) {
        h_a[i] = 2*i*i;
        h_b[i] = i ;
    }

    // Copy input arrays from host to device memory
    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);

    // Calling kernels with N blocks and one thread per block, passing
    // device pointers as parameters
    gpuAdd <<<N, 1 >>>(d_a, d_b, d_c);
    // Copy result back to host memory from device memory
    hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);
    printf("Vector addition on GPU \n");
    // Printing result on console
    for (int i = 0; i < N; i++) {
        printf("The sum of %d element is %d + %d = %d\n",
            i, h_a[i], h_b[i],h_c[i]);
    }
    // Free up memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}